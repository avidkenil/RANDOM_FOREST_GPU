
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <float.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define TRAIN_NUM 100
#define TEST_NUM 50
#define FEATURE 4
#define NUMBER_OF_CLASSES 3


#define FEAT_KEY 0
#define CUT_KEY 1
#define LEFT_KEY 2
#define RIGHT_KEY 3
#define PRED_KEY 4
#define DEPTH_KEY 5

#define NUM_FIELDS 6

#define index(i, j, N)  ((i)*(N)) + (j)
#define ixt(i, j, t, N, T) ((t)*(N)*(T)) + ((i)*(N)) + (j)
#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))
 
int countNumRows(char *filename)
{
	FILE *fp;
	int count = 0;  // Line counter (result)
	//char filename[MAX_FILE_NAME];
	char c;  // To store a character read from file
 
	// Get file name from user. The file should be
	// either in current folder or complete path should be provided
	//printf("Enter file name: ");
	//scanf("%s", filename);
 
	// Open the file
	fp = fopen(filename, "r");
 
	// Check if file exists
	if (fp == NULL)
	{
		printf("Could not open file %s", filename);
		return -1;
	}
 
	// Extract characters from file and store in character c
	for (c = getc(fp); c != EOF; c = getc(fp))
		if (c == '\n') // Increment count if this character is newline
			count = count + 1;
 
	// Close the file
	fclose(fp);
	//printf("The file %s has %d lines\n ", filename, count);
 
	return count;
}

const char* getfield(char* line, int num){
	const char* tok;
	for (tok = strtok(line, ",");
			tok && *tok;
			tok = strtok(NULL, ",\n"))
	{
		if (!--num)
			return tok;
	}
	return NULL;
}
/*
Labels for IRIS:
Iris-setosa - 0
Iris-versicolor - 1
Iris-virginica - 2
*/
void read_csv_iris(float *data, float *label, int row_count, char *filename){
	//data = (float *)malloc(row_count*4*sizeof(float));
	//label = (int *)malloc(row_count*sizeof(int));
	FILE *fp = fopen(filename,"r");
	char line[1024];
	int idx = 0;
	for(int iter = 0;iter<row_count;iter++)
	{
		fgets(line,1024,fp);
		const char *temp_field;
		for(int i=0;i<5;i++)
		{
			float temp_num;
			char *tmp = strdup(line);
			temp_field = getfield(tmp,i+1);
			if(i==4)
			{
				if(strcmp(temp_field,"Iris-setosa")==0)
				{
					label[idx] = 0;
					continue;
				}
				if(strcmp(temp_field,"Iris-versicolor")==0)
				{
					label[idx] = 1;
					continue;
				}
				if(strcmp(temp_field,"Iris-virginica")==0)
				{
					label[idx] = 2;
					continue;
				}
			}
			temp_num = atof(temp_field);
			data[idx*4 + i] = temp_num;
		}
		idx++;
		
	}
}


/* === Utils === */
int next_pow_2(int x){
	int y = 1;
	while(y < x)
		y*=2;
	return y;
}
void debug(int i){
	hipError_t e=hipGetLastError();                                 \
	if(e!=hipSuccess) {                                              \
		printf("%d Cuda failure %s:%d: '%s'\n", i, __FILE__,__LINE__,hipGetErrorString(e));    
	}
}
void copy_transpose(float* to, float* from, int h, int w){
	for(int i=0; i<h; i++){
		for(int j=0; j<w; j++){
			to[index(j, i, h)] = from[index(i, j, w)];
		}
	}
}

/* === Random Init === */
__global__ void init_random(unsigned int seed, hiprandState_t* states) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &states[tid]);
}
__device__ int draw_approx_binomial(int n, float p, hiprandState_t* state) {
	int x = (int) round(hiprand_normal(state) * n*p*(1-p) + n*p);
	return max(0, min(x, n));
}
__device__ float draw_uniform(float minimum, float maximum, hiprandState_t* state){
	return minimum + hiprand_uniform(state) * (maximum - minimum);
}

/* === Expanding tree memory === */
float* expand(float* d_trees, int num_trees, int tree_arr_length, int new_tree_arr_length){
	float *new_d_trees;
	assert(new_tree_arr_length >= tree_arr_length);

	hipMalloc((void **) &new_d_trees, num_trees * NUM_FIELDS * new_tree_arr_length *sizeof(float));
	hipMemcpy(new_d_trees, d_trees, num_trees * NUM_FIELDS * tree_arr_length *sizeof(float), hipMemcpyDeviceToDevice);
	hipFree(d_trees);
	return new_d_trees;
}
__global__ void get_max_tree_length(int* d_tree_lengths, int num_trees, int* d_max_tree_length){
	extern __shared__ int tree_length_buffer[];
	if(threadIdx.x < num_trees){
		tree_length_buffer[threadIdx.x] = d_tree_lengths[threadIdx.x];
	}else{
		tree_length_buffer[threadIdx.x] = -1;
	}
	
	for(int stride=blockDim.x/2; stride > 0; stride >>=1){
		__syncthreads();
		if(threadIdx.x < stride){
			if(tree_length_buffer[threadIdx.x + stride] > tree_length_buffer[threadIdx.x]){
				tree_length_buffer[threadIdx.x] = tree_length_buffer[threadIdx.x + stride];
			}
		}
	}
	if(threadIdx.x == 0){
	   d_max_tree_length[0] = tree_length_buffer[0];
	}
}
float* maybe_expand(float* d_trees, int num_trees, int* tree_arr_length, int* d_tree_lengths,
	                int* max_tree_length, int* d_max_tree_length){
	// I wonder if it's faster just to compute max on CPU.
	int new_tree_arr_length;
	float *new_d_trees;

	get_max_tree_length<<<1, num_trees, next_pow_2(num_trees) * sizeof(int)>>>(
		d_tree_lengths, num_trees, d_max_tree_length
	);
	hipMemcpy(max_tree_length, d_max_tree_length, sizeof(int), hipMemcpyDeviceToHost);
	// Buffer of 2 => up to 2 additions at a time
	if(*max_tree_length <= *tree_arr_length-3){
		return d_trees;
	}else{
		new_tree_arr_length = (*tree_arr_length) * 2;
        while(*max_tree_length > new_tree_arr_length-2){
            new_tree_arr_length *= 2;
        }

        printf("Expanding to %d\n", new_tree_arr_length);
        new_d_trees = expand(d_trees, num_trees, *tree_arr_length, new_tree_arr_length);
        *tree_arr_length = new_tree_arr_length;
        return new_d_trees;
	}
}

/* === Tree Initialization === */
__global__ void kernel_initialize_trees(float *d_trees, int* d_tree_lengths, int tree_arr_length){
	d_trees[ixt(0, LEFT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = 0;
	d_trees[ixt(0, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = 0;
	d_trees[ixt(0, DEPTH_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = 0;
	d_trees[ixt(0, PRED_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_tree_lengths[threadIdx.x] = 1;
}
void initialize_trees(float* d_trees, int num_trees, int tree_arr_length, int* d_tree_lengths){
	kernel_initialize_trees<<<1, num_trees>>>(d_trees, d_tree_lengths, tree_arr_length);
}
__global__ void kernel_initialize_batch_pos(int *d_batch_pos, int x_length, int num_trees){
	int i;
	for(i=threadIdx.x; i<x_length; i+=blockDim.x){
		d_batch_pos[index(blockIdx.x, i, x_length)] = 0;
	}
}
void initialize_batch_pos(int *d_batch_pos, int x_length, int num_trees, hipDeviceProp_t dev_prop){
	kernel_initialize_batch_pos<<<num_trees, dev_prop.maxThreadsPerBlock>>>(
		d_batch_pos, x_length, num_trees
	);
}
__global__ void kernel_refresh_tree_is_done(int* d_tree_lengths, int* d_tree_is_done, int tree_pos){
	// threadIdx.x = tree_id
	int is_done;
	if(tree_pos < d_tree_lengths[threadIdx.x]){
		is_done = 0;
	}else{
		is_done = 1;
	}
	d_tree_is_done[threadIdx.x] = is_done;
}
void refresh_tree_is_done(int* d_tree_lengths, int* d_tree_is_done, int tree_pos, int num_trees){
	kernel_refresh_tree_is_done<<<1, num_trees>>>(
		d_tree_lengths, d_tree_is_done, tree_pos
	);
}

/* === Tree Traversal === */
__global__ void kernel_traverse_trees(float *d_trees, float* d_x, int x_length, int num_trees, int* d_batch_pos){
	// Should optimize this. It's just a bunch of global reads.
	// Also possibly to rewrite this and batch_traverse to support a "next-step" method instead of a full 
	//   traversal while growing
	int pos, new_pos, left_right_key, x_i, tree_id;
	//Overloading x_i as tx
	x_i = threadIdx.x + blockIdx.x * blockDim.x;
	if(x_i >= x_length * num_trees) return;

	// Actually get x_i, tree_id
	tree_id = x_i % num_trees;
	x_i = x_i / num_trees;
	pos = 0;
    while(1){
        if(d_x[index(x_i, (int) d_trees[ixt(pos, FEAT_KEY, tree_id, NUM_FIELDS, TRAIN_NUM)], FEATURE)] < 
        		d_trees[ixt(pos, CUT_KEY, tree_id, NUM_FIELDS, TRAIN_NUM)]){
            left_right_key = LEFT_KEY;
        }else{
            left_right_key = RIGHT_KEY;
        }
        new_pos = (int) d_trees[ixt(pos, left_right_key, tree_id, NUM_FIELDS, TRAIN_NUM)];
        if(new_pos == pos){
            // Leaf nodes are set up to be idempotent
            break;
        }
        pos = new_pos;
    }
    d_batch_pos[index(tree_id, x_i, TRAIN_NUM)] = pos;
}
void batch_traverse_trees(float *d_tree, float *d_x, int x_length, int num_trees, int *d_batch_pos, hipDeviceProp_t dev_prop){
	int block_size, num_blocks;
 	block_size = dev_prop.maxThreadsPerBlock;
 	num_blocks = ceil(num_trees * x_length/((float) block_size));
	kernel_traverse_trees<<<num_blocks, block_size>>>(d_tree, d_x, x_length, num_trees, d_batch_pos);
}
__global__ void kernel_advance_trees(float *d_trees, float* d_x, int x_length, int tree_arr_length, int num_trees, int* d_batch_pos){
	int pos, left_right_key, x_i;
	// threadIdx.x = x_i, blockIdx.x = tree_id
	for(x_i=threadIdx.x; x_i < x_length; x_i+=blockDim.x){
		pos = d_batch_pos[index(blockIdx.x, x_i, TRAIN_NUM)];
	    if(d_x[index(x_i, (int) d_trees[ixt(pos, FEAT_KEY, blockIdx.x, NUM_FIELDS, tree_arr_length)], FEATURE)] < 
	    		d_trees[ixt(pos, CUT_KEY, blockIdx.x, NUM_FIELDS, tree_arr_length)]){
	        left_right_key = LEFT_KEY;
	    }else{
	        left_right_key = RIGHT_KEY;
	    }
	    d_batch_pos[index(blockIdx.x, x_i, TRAIN_NUM)] = (int) d_trees[ixt(pos, left_right_key, blockIdx.x, NUM_FIELDS, tree_arr_length)];
	}
}
void batch_advance_trees(float *d_tree, float *d_x, int x_length, int tree_arr_length, int num_trees, int *d_batch_pos, 
						 hipDeviceProp_t dev_prop){
	kernel_advance_trees<<<num_trees, dev_prop.maxThreadsPerBlock>>>(
		d_tree, d_x, x_length, tree_arr_length, num_trees, d_batch_pos
	);
}

/* === Node termination === */
__global__ void kernel_check_node_termination(
			float* d_trees, int tree_arr_length,
			float* d_y, int* d_batch_pos, int tree_pos, 
			int* d_is_branch_node, int* d_tree_is_done
		){
	// threadIdx.x = tree_id
	int i, base_y, new_y, is_branch_node;

	// If tree is done, it's never a branch node
	if(d_tree_is_done[threadIdx.x]==1){
		d_is_branch_node[threadIdx.x] = 0;
		return;
	}

	// Check for non-unique Y
	base_y = -1;
	is_branch_node = 0;
	for(i=1; i<TRAIN_NUM; i++){
		if(d_batch_pos[index(threadIdx.x, i, TRAIN_NUM)] == tree_pos){
			new_y = d_y[i];
			if(base_y == -1){
				base_y = new_y;
			}else if(base_y != new_y){
				is_branch_node = 1;
				break;
			}
		}
	}
	d_is_branch_node[threadIdx.x] = is_branch_node;

	if(base_y==-1){
		printf("ERROR ERROR ERROR EMPTY 1TREE %d\n", threadIdx.x);
		printf("ERROR ERROR ERROR EMPTY 2TREE %d\n", threadIdx.x);
		printf("ERROR ERROR ERROR EMPTY 2TREE %d\n", threadIdx.x);
	}

	if(!is_branch_node){
		d_trees[ixt(tree_pos, PRED_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = base_y;
	}
}
void check_node_termination(
			float* d_trees, int tree_arr_length,
			float* d_y, int* d_batch_pos, int tree_pos, 
			int* d_is_branch_node, int* d_tree_is_done,
			int num_trees
		){
	kernel_check_node_termination<<<1, num_trees>>>(
		d_trees, tree_arr_length, 
		d_y, d_batch_pos, tree_pos,
		d_is_branch_node, d_tree_is_done
	);
}

/* === Valid features === */
__global__ void kernel_collect_min_max(float* d_x_T, int* d_batch_pos, int desired_pos, int num_trees, 
									   int x_length, float* d_min_max_buffer){
	extern __shared__ float shared_min_max[]; // threadIdx.x * 2
	// Ripe for optimization.
	// threadIdx.x = x_i++, blockIdx.x = tree_id, feat = blockIdx.y
	int x_i;
	float minimum, maximum, val;

	minimum = FLT_MAX;
	maximum = -FLT_MAX;
	for(x_i=threadIdx.x; x_i < x_length; x_i+=blockDim.x){
		if(d_batch_pos[index(blockIdx.x, x_i, x_length)] == desired_pos){
			val = d_x_T[index(blockIdx.y, x_i, TRAIN_NUM)];
			if(val < minimum){
				minimum = val;
			}
			if(val > maximum){
				maximum = val;
			}
		}
	}
	shared_min_max[index(threadIdx.x, 0, 2)] = minimum;
	shared_min_max[index(threadIdx.x, 1, 2)] = maximum;

	for(int stride=blockDim.x/2; stride > 0; stride >>=1){
		__syncthreads();
		if(threadIdx.x < stride){
			if(shared_min_max[index(threadIdx.x + stride, 0, 2)] < shared_min_max[index(threadIdx.x, 0, 2)]){
				shared_min_max[index(threadIdx.x, 0, 2)] = shared_min_max[index(threadIdx.x + stride, 0, 2)];
			}
			if(shared_min_max[index(threadIdx.x + stride, 1, 2)] > shared_min_max[index(threadIdx.x, 1, 2)]){
				shared_min_max[index(threadIdx.x, 1, 2)] = shared_min_max[index(threadIdx.x + stride, 1, 2)];
			}
		}
	}
	if(threadIdx.x==0){
		d_min_max_buffer[ixt(blockIdx.y, 0, blockIdx.x, 2, FEATURE)] = shared_min_max[index(0, 0, 2)];
		d_min_max_buffer[ixt(blockIdx.y, 1, blockIdx.x, 2, FEATURE)] = shared_min_max[index(0, 1, 2)];
	}
}
void collect_min_max(float* d_x_T, int* d_batch_pos, int desired_pos, int num_trees, int x_length,
					 float* d_min_max_buffer, hipDeviceProp_t dev_prop){
	// Ripe for optimization.
	dim3 grid(num_trees, FEATURE);
	kernel_collect_min_max<<<grid, 64, 64 * sizeof(int) * 2>>>(
		d_x_T, d_batch_pos, desired_pos, num_trees, x_length, d_min_max_buffer
	);	
}
__global__ void kernel_collect_num_valid_feat(int* d_num_valid_feat, float* d_min_max_buffer, int num_trees){
	extern __shared__ int shared_num_valid_feat_buffer[];
	// blockIdx.x = tree_id
	int sub_num_valid_feat, feat_i;
	sub_num_valid_feat = 0;
	for(feat_i=threadIdx.x; feat_i<FEATURE; feat_i+=blockDim.x){
		if(d_min_max_buffer[ixt(feat_i, 0, blockIdx.x, 2, FEATURE)] != 
			d_min_max_buffer[ixt(feat_i, 1, blockIdx.x, 2, FEATURE)]
			){
			sub_num_valid_feat++;
		}
	}
	shared_num_valid_feat_buffer[threadIdx.x] = sub_num_valid_feat;
	for(int stride=blockDim.x/2; stride > 0; stride >>=1){
		__syncthreads();
		if(threadIdx.x < stride){
			shared_num_valid_feat_buffer[threadIdx.x] += shared_num_valid_feat_buffer[threadIdx.x + stride];
		}
	}
	if(threadIdx.x == 0){
	   d_num_valid_feat[blockIdx.x] = shared_num_valid_feat_buffer[0];
	}
}
void collect_num_valid_feat(int* d_num_valid_feat, float* d_min_max_buffer, int num_trees, hipDeviceProp_t dev_prop){
	// Ripe for optimization
	int block_size = MIN(dev_prop.maxThreadsPerBlock, next_pow_2(FEATURE)); // Copy this to other places too
	kernel_collect_num_valid_feat<<<num_trees, block_size, block_size * sizeof(int)>>>(
		d_num_valid_feat, d_min_max_buffer, num_trees
	);
}

/* === Populate Random Features === */
__global__ void kernel_depopulate_valid_feat_idx(int* d_random_feats, int num_trees, int feat_per_node){
	int t;
	for(t=0; t<num_trees; t++){
		//-1 means fill-forward
		d_random_feats[index(t, threadIdx.x, feat_per_node)] = -1;
	}
}
__global__ void kernel_populate_valid_feat_idx(int* d_random_feats, int* d_num_valid_feat, int feat_per_node, 
	                         				   hiprandState_t* curand_states){
	// threadIdx.x = tree_id
	int k, idx, draw, num_valid_feat;
	idx = 0;
	num_valid_feat = d_num_valid_feat[threadIdx.x];
	for(k=0; k<(num_valid_feat-1); k++){
		draw = draw_approx_binomial(feat_per_node-idx, 1./(num_valid_feat-k), curand_states + threadIdx.x);
		if(draw > 0){
			d_random_feats[index(threadIdx.x, idx, feat_per_node)] = k;
		}
		idx += draw;
		if(idx >= feat_per_node){
			return;
		}
	}
	if(idx < feat_per_node){
		d_random_feats[index(threadIdx.x, idx, feat_per_node)] = k;
	}
}
__global__ void kernel_populate_feat_cut(int* d_random_feats, float* d_random_cuts,
										 float* d_min_max_buffer, int feat_per_node,
										 int num_trees, hiprandState_t* curand_states){
	// threadIdx.x = tree_id
	int feat_i, feat_idx, feat_idx_idx, valid_feats_seen, buffer;
	float minimum, maximum;
	feat_idx = -1; // First element will overwrite
	feat_idx_idx = 0; // Parallel construction
	valid_feats_seen = 0;
	for(feat_i=0; feat_i < FEATURE; feat_i++){
		minimum = d_min_max_buffer[ixt(feat_i, 0, threadIdx.x, 2, FEATURE)];
		maximum = d_min_max_buffer[ixt(feat_i, 1, threadIdx.x, 2, FEATURE)];
		if(minimum!=maximum){
			while(1){
				buffer = d_random_feats[index(threadIdx.x, feat_idx_idx, feat_per_node)];
				if(buffer != -1){
					feat_idx = buffer;
				}
				if(feat_idx==valid_feats_seen){
					d_random_feats[index(threadIdx.x, feat_idx_idx, feat_per_node)] = feat_i;
					d_random_cuts[index(threadIdx.x, feat_idx_idx, feat_per_node)] = draw_uniform(minimum, maximum, curand_states+threadIdx.x);
				}else{
					break;
				}
				feat_idx_idx++;
				if(feat_idx_idx >= feat_per_node){
					return;
				}
			}
		}
		valid_feats_seen++;
	}
}
void populate_valid_feat_idx(int* d_random_feats, int* d_num_valid_feat, int feat_per_node, int num_trees,
							 hiprandState_t* curand_states){
	kernel_depopulate_valid_feat_idx<<<1, feat_per_node>>>(d_random_feats, num_trees, feat_per_node);
	kernel_populate_valid_feat_idx<<<1, num_trees>>>(
		d_random_feats, d_num_valid_feat, feat_per_node, curand_states
	);
}
void populate_feat_cut(int* d_random_feats, float* d_random_cuts,
	 				   float* d_min_max_buffer, int feat_per_node,
	 				   int num_trees, hiprandState_t* curand_states){
	kernel_populate_feat_cut<<<1, num_trees>>>(
		d_random_feats, d_random_cuts, d_min_max_buffer, feat_per_node, num_trees, curand_states
	);
}

/* === Count Classes === */
__global__ void kernel_populate_class_counts(
		float* d_x, float* d_y, int* d_class_counts_a, int* d_class_counts_b, 
		int* d_random_feats, float* d_random_cuts,
		int* d_batch_pos, int tree_pos,
		int num_trees, int feat_per_node
	){
	// Naive version
	// threadIdx.x = tree_id, blockIdx.x = rand_feat_i
	int i, y, feat;
	float cut;
	feat = d_random_feats[index(threadIdx.x, blockIdx.x, feat_per_node)];
	cut = d_random_cuts[index(threadIdx.x, blockIdx.x, feat_per_node)];
	for(i=0; i<NUMBER_OF_CLASSES; i++){
		//tree node class
		d_class_counts_a[ixt(threadIdx.x, blockIdx.x, i, feat_per_node, num_trees)] = 0;
		d_class_counts_b[ixt(threadIdx.x, blockIdx.x, i, feat_per_node, num_trees)] = 0;
	}
	for(i=0; i<TRAIN_NUM; i++){
		if(d_batch_pos[index(threadIdx.x, i, TRAIN_NUM)]==tree_pos){
			y = (int) d_y[i];
			if(d_x[index(i, feat, FEATURE)] < cut){
				d_class_counts_a[ixt(threadIdx.x, blockIdx.x, y, feat_per_node, num_trees)]++;
			}else{
				d_class_counts_b[ixt(threadIdx.x, blockIdx.x, y, feat_per_node, num_trees)]++;
			}
		}
	}
}
void populate_class_counts(
		float* d_x, float* d_y, int* d_class_counts_a, int* d_class_counts_b, 
		int* d_random_feats, float* d_random_cuts,
		int* d_batch_pos, int tree_pos,
		int num_trees, int feat_per_node
	){
	// Naive version
	kernel_populate_class_counts<<<feat_per_node, num_trees>>>(
		d_x, d_y, d_class_counts_a, d_class_counts_b, 
		d_random_feats, d_random_cuts,
		d_batch_pos, tree_pos,
		num_trees, feat_per_node
	);
}

/* === Place Best Features/Cuts === */
__global__ void kernel_place_best_feat_cuts(
		int* d_class_counts_a, int* d_class_counts_b, 
		int* d_random_feats, float* d_random_cuts,
		int* d_best_feats, float* d_best_cuts,
		int feat_per_node, int num_trees
	){
	// Naive version => Can move class_counts into shared memory
	// threadIdx.x = tree_id
	int i, k;
    float best_improvement, best_cut, proxy_improvement;
    int best_feat;
    int total_a, total_b;
    float impurity_a, impurity_b;

    best_improvement = -FLT_MAX;
    best_feat = -1;
    best_cut = 0;
	for(i=0; i<feat_per_node; i++){
        total_a = 0;
        total_b = 0;
        impurity_a = 1;
        impurity_b = 1;
        for(k=0; k<NUMBER_OF_CLASSES; k++){
            total_a += d_class_counts_a[ixt(threadIdx.x, i, k, feat_per_node, num_trees)];
            total_b += d_class_counts_b[ixt(threadIdx.x, i, k, feat_per_node, num_trees)];
        }
        for(k=0; k<NUMBER_OF_CLASSES; k++){
            impurity_a -= pow(((float) d_class_counts_a[ixt(threadIdx.x, i, k, feat_per_node, num_trees)]) / total_a, 2);
            impurity_b -= pow(((float) d_class_counts_b[ixt(threadIdx.x, i, k, feat_per_node, num_trees)]) / total_b, 2);
        }
        proxy_improvement = - total_a * impurity_a - total_b * impurity_b;
        if(proxy_improvement > best_improvement){
            best_feat = d_random_feats[index(threadIdx.x, i, feat_per_node)];
            best_cut = d_random_cuts[index(threadIdx.x, i, feat_per_node)];
            best_improvement = proxy_improvement;
        }
	}
	d_best_feats[threadIdx.x] = best_feat;
	d_best_cuts[threadIdx.x] = best_cut;
}
void place_best_feat_cuts(
		int* d_class_counts_a, int* d_class_counts_b, 
		int* d_random_feats, float* d_random_cuts,
		int* d_best_feats, float* d_best_cuts,
		int feat_per_node, int num_trees
	){
	// Naive version
	kernel_place_best_feat_cuts<<<1, num_trees>>>(
		d_class_counts_a, d_class_counts_b, 
		d_random_feats, d_random_cuts,
		d_best_feats, d_best_cuts,
		feat_per_node, num_trees
	);
}

/* === Update Trees === */
__global__ void kernel_update_trees(
			float* d_trees, int* d_tree_lengths, int tree_pos,
			int* d_best_feats, float* d_best_cuts, int tree_arr_length
		){
	// Naive version
	// threadIdx.x = tree_id
	int left_child_pos, right_child_pos, tree_length;
	tree_length = d_tree_lengths[threadIdx.x];
	left_child_pos = tree_length;
	right_child_pos = tree_length + 1;

	// Update tree nodes
	d_trees[ixt(tree_pos, LEFT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = left_child_pos;
	d_trees[ixt(tree_pos, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = right_child_pos;
	d_trees[ixt(tree_pos, FEAT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = d_best_feats[threadIdx.x];
	d_trees[ixt(tree_pos, CUT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = d_best_cuts[threadIdx.x];
	d_tree_lengths[threadIdx.x] += 2;

	// Prefill child nodes
	d_trees[ixt(left_child_pos, LEFT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = left_child_pos;
	d_trees[ixt(left_child_pos, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = left_child_pos;
	d_trees[ixt(left_child_pos, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = \
		d_trees[ixt(tree_pos, DEPTH_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] + 1;
	d_trees[ixt(left_child_pos, FEAT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_trees[ixt(left_child_pos, CUT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_trees[ixt(left_child_pos, PRED_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;

	d_trees[ixt(right_child_pos, LEFT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = left_child_pos;
	d_trees[ixt(right_child_pos, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = left_child_pos;
	d_trees[ixt(right_child_pos, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = \
		d_trees[ixt(tree_pos, DEPTH_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] + 1;
	d_trees[ixt(right_child_pos, FEAT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_trees[ixt(right_child_pos, CUT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_trees[ixt(right_child_pos, PRED_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
}
void update_trees(
			float* d_trees, int* d_tree_lengths, int tree_pos,
			int* d_best_feats, float* d_best_cuts, int tree_arr_length,
				int num_trees
		){
	kernel_update_trees<<<1, num_trees>>>(
		d_trees, d_tree_lengths, tree_pos,
		d_best_feats, d_best_cuts, tree_arr_length
	);
}


int main(int argc,char *argv[])
{
	float *dataset_train,*dataset_test;
	float *labels_train,*labels_test;
	dataset_train = (float *)malloc(FEATURE * TRAIN_NUM*sizeof(float));
	labels_train = (float *)malloc(TRAIN_NUM*sizeof(float));
	dataset_test = (float *)malloc(FEATURE * TEST_NUM*sizeof(float));
	labels_test = (float *)malloc(TEST_NUM*sizeof(float));
	char file_train_set[] = "data/iris_train.data";
	char file_test_set[] = "data/iris_test.data";
	read_csv_iris(dataset_train,labels_train,TRAIN_NUM,file_train_set);
	read_csv_iris(dataset_test,labels_test,TEST_NUM,file_test_set);
	
	float *dataset_train_T;
	dataset_train_T = (float *)malloc(TRAIN_NUM * FEATURE * sizeof(float));
	copy_transpose(dataset_train_T, dataset_train, TRAIN_NUM, FEATURE);

	float *trees, *d_trees;
	int *tree_arr_length;
	int *tree_lengths, *d_tree_lengths;
	int *max_tree_length, *d_max_tree_length;
	int feat_per_node;
	int *num_valid_feat, *d_num_valid_feat;
	int tree_pos;
	int *batch_pos, *d_batch_pos; // NUM_TREES * TRAIN_NUM
	int *is_branch_node, *d_is_branch_node;
	int *tree_is_done, *d_tree_is_done;
	float *min_max_buffer, *d_min_max_buffer;
	int *random_feats, *d_random_feats;
	float *random_cuts, *d_random_cuts;
	int *class_counts_a, *class_counts_b;
	int *d_class_counts_a, *d_class_counts_b;
	int *best_feats, *d_best_feats;
	float *best_cuts, *d_best_cuts;
	int prev_depth, max_depth;
	float *d_x, *d_y;
	float *d_x_T;
	hiprandState_t* curand_states;

	int num_trees;
	num_trees = 5;
	// Assumption: num_trees < maxNumBlocks, maxThreadsPerBlock
	srand(2);

	tree_arr_length = (int *)malloc(sizeof(int));
	tree_lengths = (int *)malloc(num_trees * sizeof(int));
	*tree_arr_length = 1024;
	max_tree_length = (int *)malloc(sizeof(int));

	feat_per_node = (int) ceil(sqrt(FEATURE));

	trees = (float *)malloc(num_trees * NUM_FIELDS * (*tree_arr_length) *sizeof(float));
	batch_pos = (int *)malloc(num_trees * TRAIN_NUM *sizeof(float));
	is_branch_node = (int *)malloc(num_trees * sizeof(int));
	tree_is_done = (int *)malloc(num_trees * sizeof(int));
	min_max_buffer = (float *)malloc(num_trees * FEATURE * 2 *sizeof(float));
	
	num_valid_feat = (int *)malloc(num_trees * sizeof(int));
	random_feats = (int *)malloc(num_trees * feat_per_node * sizeof(int));
	random_cuts = (float *)malloc(num_trees * feat_per_node * sizeof(float));

	best_feats = (int *)malloc(num_trees * sizeof(int));
	best_cuts = (float *)malloc(num_trees * sizeof(float));

	class_counts_a = (int *)malloc(num_trees * feat_per_node * NUMBER_OF_CLASSES *sizeof(int));
	class_counts_b = (int *)malloc(num_trees * feat_per_node * NUMBER_OF_CLASSES *sizeof(int));
	hipDeviceProp_t dev_prop;
	hipGetDeviceProperties(&dev_prop, 0);
	hipMalloc((void **) &d_trees, num_trees * NUM_FIELDS * (*tree_arr_length) *sizeof(float));
	hipMalloc((void **) &d_tree_lengths, num_trees * sizeof(int));
	hipMalloc((void **) &d_max_tree_length, sizeof(int));
	hipMalloc((void **) &d_batch_pos, num_trees * TRAIN_NUM *sizeof(float));
	hipMalloc((void **) &d_is_branch_node, num_trees * sizeof(int));
	hipMalloc((void **) &d_tree_is_done, num_trees * sizeof(int));
	hipMalloc((void **) &d_min_max_buffer, num_trees * FEATURE * 2 *sizeof(float));
	hipMalloc((void **) &d_num_valid_feat, num_trees *sizeof(int));
	hipMalloc((void **) &d_random_feats, num_trees * feat_per_node * sizeof(int));
	hipMalloc((void **) &d_random_cuts, num_trees * feat_per_node * sizeof(float));
	hipMalloc((void **) &d_best_feats, num_trees * sizeof(int));
	hipMalloc((void **) &d_best_cuts, num_trees * sizeof(float));
	hipMalloc((void **) &d_class_counts_a, num_trees * feat_per_node * NUMBER_OF_CLASSES *sizeof(int));
	hipMalloc((void **) &d_class_counts_b, num_trees * feat_per_node * NUMBER_OF_CLASSES *sizeof(int));
	hipMalloc((void **) &d_x, TRAIN_NUM * FEATURE *sizeof(float));
	hipMalloc((void **) &d_y, TRAIN_NUM *sizeof(float));
	hipMalloc((void **) &d_x_T, TRAIN_NUM * FEATURE *sizeof(float));
	hipMemcpy(d_x, dataset_train, TRAIN_NUM * FEATURE *sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, labels_train, TRAIN_NUM *sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x_T, dataset_train_T, TRAIN_NUM * FEATURE *sizeof(float), hipMemcpyHostToDevice);


	hipMalloc((void**) &curand_states, num_trees * sizeof(hiprandState));
	init_random<<<1, num_trees>>>(1337, curand_states);

	initialize_trees(d_trees, num_trees, *tree_arr_length, d_tree_lengths);
	initialize_batch_pos(d_batch_pos, TRAIN_NUM, num_trees, dev_prop);

	for(tree_pos=0; tree_pos<2; tree_pos++){
		printf("* ================== TREE POS -[ %d ]- ================== *\n", tree_pos);
		refresh_tree_is_done(d_tree_lengths, d_tree_is_done, tree_pos, num_trees);
		maybe_expand(d_trees, num_trees, tree_arr_length, d_tree_lengths, max_tree_length, d_max_tree_length);
		batch_advance_trees(d_trees, d_x, TRAIN_NUM, *tree_arr_length, num_trees, d_batch_pos, dev_prop);
		check_node_termination(
			d_trees, *tree_arr_length, 
			d_y, d_batch_pos, tree_pos,
			d_is_branch_node, d_tree_is_done,
			num_trees
		);

		// ^^
		hipMemcpy(is_branch_node, d_is_branch_node, num_trees * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(tree_is_done, d_tree_is_done, num_trees * sizeof(int), hipMemcpyDeviceToHost);
		printf("TREE IS DONE  : ");
		for(int i=0; i<num_trees; i++){printf("%d ", tree_is_done[i]);};printf("\n");
		printf("IS BRANCH NODE: ");
		for(int i=0; i<num_trees; i++){printf("%d ", is_branch_node[i]);};printf("\n");
		// VV

		/*
		cudaMemcpy(batch_pos, d_batch_pos, num_trees * TRAIN_NUM * sizeof(float), cudaMemcpyDeviceToHost);
		for(int i=0; i<num_trees; i++){
			for(int j=0; j<TRAIN_NUM; j++){
				printf("%d ", batch_pos[index(i, j, TRAIN_NUM)]);
			}
			printf("\n");
		}
		*/

		collect_min_max(d_x_T, d_batch_pos, tree_pos, num_trees, TRAIN_NUM,
						d_min_max_buffer, dev_prop);
		collect_num_valid_feat(
			d_num_valid_feat, d_min_max_buffer, num_trees, dev_prop
		);
		populate_valid_feat_idx(d_random_feats, d_num_valid_feat, feat_per_node, num_trees, curand_states);

		// AAAA
		/*
		cudaMemcpy(random_feats, d_random_feats, num_trees * feat_per_node * sizeof(int), cudaMemcpyDeviceToHost);
		for(int i=0; i<num_trees; i++){
			printf("T=%d:  ", i);
			for(int j=0; j<feat_per_node; j++){
				printf("%d(%d)  ", random_feats[index(i, j, feat_per_node)], index(i, j, feat_per_node));
			}
			printf("\n");
		}*/
		// ZZZZ

		populate_feat_cut(
			d_random_feats, d_random_cuts, d_min_max_buffer, feat_per_node, num_trees, curand_states
		);
		populate_class_counts(
			d_x, d_y, d_class_counts_a, d_class_counts_b, 
			d_random_feats, d_random_cuts,
			d_batch_pos, tree_pos,
			num_trees, feat_per_node
		);
		place_best_feat_cuts(
			d_class_counts_a, d_class_counts_b, 
			d_random_feats, d_random_cuts,
			d_best_feats, d_best_cuts,
			feat_per_node, num_trees
		);
		update_trees(
			d_trees, d_tree_lengths, tree_pos,
			d_best_feats, d_best_cuts, *tree_arr_length,
			num_trees
		);

		hipMemcpy(random_feats, d_random_feats, num_trees * feat_per_node * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(random_cuts, d_random_cuts, num_trees * feat_per_node * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(class_counts_a, d_class_counts_a, num_trees * feat_per_node * NUMBER_OF_CLASSES *sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(class_counts_b, d_class_counts_b, num_trees * feat_per_node * NUMBER_OF_CLASSES *sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(best_feats, d_best_feats, num_trees *  sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(best_cuts, d_best_cuts, num_trees *  sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<num_trees; i++){
			printf("T=%d\n", i);
			for(int j=0; j<feat_per_node; j++){
				printf("  J=%d  @ %d---%f\n", j, random_feats[index(i, j, feat_per_node)], random_cuts[index(i, j, feat_per_node)]);
				printf("    ");
				for(int k=0; k<NUMBER_OF_CLASSES; k++){
					printf(" %d", class_counts_a[ixt(i, j, k, feat_per_node, num_trees)]);
				}
				printf("\n");
				printf("    ");
				for(int k=0; k<NUMBER_OF_CLASSES; k++){
					printf(" %d", class_counts_b[ixt(i, j, k, feat_per_node, num_trees)]);
				}
				printf("\n");
			}
			printf("\n");
		}
		for(int i=0; i<num_trees; i++){
			printf("T=%d ==> %d/%f\n", i, best_feats[i], best_cuts[i]);
		}

	}

	/*
	for(int i=0; i<num_trees; i++){
		for(int j=0; j<feat_per_node; j++){
			printf("  %d %d %f \n", j, random_feats[index(i, j, feat_per_node)],
				                       random_cuts[index(i, j, feat_per_node)]);
		}
		printf("\n");
	}
	printf("%d\n", feat_per_node);
	*/


		/*
			TO DO:
				- Expanding is broken
				- Check 2nd level filter
				- Implement terminal nodes
				- Randomness might be broken
		*/

	printf("\n");
	debug(0);
}